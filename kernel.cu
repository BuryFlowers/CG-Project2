#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/extrema.h>
#include <hip/hip_runtime.h>
//#include <hip/device_functions.h>
//#include "sm_20_atomic_functions.h"
#include "book.h"
#include "glm/glm.hpp"
#include "device_vector.cuh"
#include "device_triangle.cuh"
#include "ray.h"
#include "tracing.h"
#include "material.h"

__device__ float d_dot(d_vec3 a, d_vec3 b) {

	return a.x * b.x + a.y * b.y + a.z * b.z;

}

__device__ float d_fabs(float a) {

	if (a < 0) return a * -1.0f;
	return a;

}

__device__ d_vec3 d_cross(d_vec3 a, d_vec3 b) {

	return d_vec3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);

}

__global__ void IntersectionCheckKernel(d_vec3 start, d_vec3 direction, d_triangle* triangles, int num, int* id, float* resultT, d_vec3* weights, int* d_lock) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < num) {

		d_triangle T = triangles[index];
		d_vec3 planeNormal;
		planeNormal = d_cross(T.v[1] - T.v[0], T.v[2] - T.v[0]);
		float area = planeNormal.length();
		planeNormal.normalize();

		if (d_fabs(d_dot(direction, planeNormal)) < 1e-5) return;
		if (area == 0) return;

		float t = (d_dot(T.v[1], planeNormal) - d_dot(start, planeNormal)) / d_dot(direction, planeNormal);
		//printf("ID:%d A:%.2lf N:%.2lf %.2lf %.2lf T:%.2lf\n", index, area, planeNormal.x, planeNormal.y, planeNormal.z, t);
		//printf("ID:%d\n", index);
		if (t < 1e-3) return;

		d_vec3 p = start + direction * t;
		d_vec3 t1 = d_cross(T.v[0] - p, T.v[1] - p);
		d_vec3 t2 = d_cross(T.v[1] - p, T.v[2] - p);
		d_vec3 t3 = d_cross(T.v[2] - p, T.v[0] - p);

		if (d_dot(t1, t2) < 0) return;
		if (d_dot(t1, t3) < 0) return;
		if (d_dot(t2, t3) < 0) return;

		float w0, w1, w2;

		w0 = t2.length() / area;
		w1 = t3.length() / area;
		w2 = 1 - w0 - w1;

		//printf("%d\n", *d_lock);
		while (true) {

			if (atomicExch(d_lock, 0) != 0) {

				//printf("%d in! %d\n", index, *d_lock);
				if (*id == -1) {

					*id = index;
					*resultT = t;
					*weights = d_vec3(w0, w1, w2);

				}
				else if (t < *resultT) {

					*id = index;
					*resultT = t;
					*weights = d_vec3(w0, w1, w2);

				}

				atomicAdd(d_lock, 1);
				//printf("%d out! %d\n", index, *d_lock);
				break;

			}

			//printf("%d wait! %d\n", index, *d_lock);

		}

	}

}

extern int triangleNum;
extern d_triangle* d_triangles;
extern d_triangle* h_triangles;
extern int* d_id;
extern float* d_T;
extern d_vec3* d_weight;
extern int* d_lock;
extern Material* h_mat;
extern int* h_id;
extern float* h_T;
extern d_vec3* h_weight;
extern int* h_lock;

bool CUDAIntersectionCheck(Ray r, float& t, IntersectionPoint& IP) {

	hipEvent_t start, stop;
	// capture the start time
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	*h_lock = 1;
	*h_T = 1e10;
	*h_id = -1;

	if (hipMemcpy(d_id, h_id, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {

		printf("[Error]Failed to copy id to GPU\n");
		exit(-1);

	}

	if (hipMemcpy(d_lock, h_lock, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {

		printf("[Error]Failed to copy locker to GPU\n");
		exit(-1);

	}

	if (hipMemcpy(d_T, h_T, sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {

		printf("[Error]Failed to copy t to GPU\n");
		exit(-1);

	}

	int threads = 512;
	int blocks = (triangleNum + threads) / threads;
	d_vec3 origin = d_vec3(r.origin().x, r.origin().y, r.origin().z);
	d_vec3 direction = d_vec3(r.direction().x, r.direction().y, r.direction().z);

	IntersectionCheckKernel <<<blocks, threads >>> (origin, direction, d_triangles, triangleNum, d_id, d_T, d_weight, d_lock);

	if (hipMemcpy(h_id, d_id, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {

		printf("[Error]Failed to send flags from GPU!");
		exit(-1);

	}

	if (hipMemcpy(h_T, d_T, sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) {

		printf("[Error]Failed to send Ts from GPU!");
		exit(-1);

	}

	if (hipMemcpy(h_weight, d_weight, sizeof(d_vec3), hipMemcpyDeviceToHost) != hipSuccess) {

		printf("[Error]Failed to send weights from GPU!");
		exit(-1);

	}

	// get stop time, and display the timing results
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float   elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
		start, stop));
	printf("GPU Time to compute:  %3.1f ns\n", elapsedTime * 1e6);

	if (*h_id == -1) return false;

	vec3 weight = vec3(h_weight->x, h_weight->y, h_weight->z);
	d_triangle T = h_triangles[*h_id];
	d_vec3 normal;
	d_vec2 uv;
	IP.mat = &h_mat[*h_id];
	IP.p = r.pointAt(*h_T);
	normal = T.n[0] * weight.x + T.n[1] * weight.y + T.n[2] * weight.z;
	IP.n = vec3(normal.x, normal.y, normal.z);
	uv = T.uv[0] * weight.x + T.uv[1] * weight.y + T.uv[2] * weight.z;
	IP.uv = vec2(uv.x, uv.y);

	return true;

}